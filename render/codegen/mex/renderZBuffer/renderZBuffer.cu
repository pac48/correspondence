#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// renderZBuffer.cu
//
// Code generation for function 'renderZBuffer'
//

// Include files
#include "renderZBuffer.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "hip/hip_math_constants.h"

// Function Declarations
static __global__ void renderZBuffer_kernel1(
    const real32_T allColor_data[], const real32_T allVertsCamera_data[],
    const real32_T allVerts2d_data[], const real32_T boundingBox_data[],
    const real32_T distances_data[], const real32_T idxAll_data[],
    const int32_T idxAll_size[3], const real32_T tree_data[],
    real_T vertInds[262144], real32_T im[786432]);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void renderZBuffer_kernel1(
    const real32_T allColor_data[], const real32_T allVertsCamera_data[],
    const real32_T allVerts2d_data[], const real32_T boundingBox_data[],
    const real32_T distances_data[], const real32_T idxAll_data[],
    const int32_T idxAll_size[3], const real32_T tree_data[],
    real_T vertInds[262144], real32_T im[786432])
{
  uint64_T threadId;
  int32_T px;
  int32_T py;
  real32_T b_colors[9];
  real32_T b_verts2d[6];
  real32_T colors[6];
  real32_T verts2d[6];
  real32_T A[4];
  real32_T point[3];
  real32_T deltas[2];
  real32_T point2d[2];
  real32_T u[2];
  int8_T distInds[2];
  boolean_T x[2];
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  py = static_cast<int32_T>(threadId % 512ULL);
  px = static_cast<int32_T>((threadId - static_cast<uint64_T>(py)) / 512ULL);
  if ((static_cast<int32_T>(px < 512)) && (static_cast<int32_T>(py < 512))) {
    int32_T binInd;
    int32_T faceInd;
    real32_T depth;
    real32_T pixelX;
    real32_T pixelY;
    faceInd = -1;
    pixelX = 2.0F * ((static_cast<real32_T>(px) + 1.0F) - 256.0F) / 512.0F;
    pixelY = 2.0F * ((static_cast<real32_T>(py) + 1.0F) - 256.0F) / 512.0F;
    //  pixelX = 2*(px - resX/2)/(resX);
    //  pixelY = 2*(py - resY/2)/(resY);
    depth = HIP_INF_F;
    //  depth = (inf);
    point[0] = 0.0F;
    point[1] = 0.0F;
    point[2] = 0.0F;
    //  color = coder.nullcopy(zeros(3, 1));
    binInd = 1;
    while (binInd != 0) {
      int32_T b_childInd;
      int32_T binInd2;
      int32_T childInd;
      real32_T absxk;
      real32_T t;
      binInd2 = binInd;
      t = tree_data[61 * (binInd - 1) + 5] +
          (tree_data[61 * (binInd - 1) + 6] -
           tree_data[61 * (binInd - 1) + 5]) /
              2.0F;
      absxk = tree_data[61 * (binInd - 1) + 7] +
              (tree_data[61 * (binInd - 1) + 8] -
               tree_data[61 * (binInd - 1) + 7]) /
                  2.0F;
      childInd = ((static_cast<int32_T>(pixelY < absxk) << 1) +
                  static_cast<int32_T>(pixelX > t)) +
                 1;
      if (childInd - 1 == 0) {
        b_childInd = 0;
      } else {
        b_childInd = static_cast<int32_T>(
            fmodf(static_cast<real32_T>(childInd) - 1.0F, 2.0F));
      }
      distInds[0] = static_cast<int8_T>(b_childInd + 1);
      distInds[1] = static_cast<int8_T>(
          4 - static_cast<int32_T>(
                  floorf((static_cast<real32_T>(childInd) - 1.0F) / 2.0F)));
      deltas[0] = fabsf(pixelX - t);
      deltas[1] = fabsf(pixelY - absxk);
      //  for k = 1:4
      //  distInd1 = mod(k-1, 2)+1;
      //      distInd2 = floor((k-1)/2)+3;
      //      [distInd1 distInd2]
      //  end
      //  child1: dist 1 4
      //  child2: dist 2 4
      //  child3: dist 1 3
      //  child4: dist 2 3
      while (binInd2 != 0) {
        //  loop through all overflow
        for (int32_T dInd{0}; dInd < 2; dInd++) {
          b_childInd = 0;
          int32_T exitg1;
          do {
            int32_T r1;
            exitg1 = 0;
            t = roundf(tree_data[61 * (binInd2 - 1) + 9]);
            if (t < 2.14748365E+9F) {
              if (t >= -2.14748365E+9F) {
                r1 = static_cast<int32_T>(t);
              } else {
                r1 = MIN_int32_T;
              }
            } else if (t >= 2.14748365E+9F) {
              r1 = MAX_int32_T;
            } else {
              r1 = 0;
            }
            if (b_childInd < r1) {
              int32_T i;
              int32_T r2;
              b_childInd++;
              t = roundf(
                  idxAll_data[((b_childInd + 50 * (binInd2 - 1)) +
                               50 * idxAll_size[1] *
                                   (static_cast<int32_T>(distInds[dInd]) - 1)) -
                              1]);
              if (t < 2.14748365E+9F) {
                if (t >= -2.14748365E+9F) {
                  r2 = static_cast<int32_T>(t);
                } else {
                  r2 = MIN_int32_T;
                }
              } else if (t >= 2.14748365E+9F) {
                r2 = MAX_int32_T;
              } else {
                r2 = 0;
              }
              if (r2 >= -2147483647) {
                r2--;
              }
              if (r2 > 2147483635) {
                r2 = MAX_int32_T;
              } else {
                r2 += 12;
              }
              t = roundf(tree_data[(r2 + 61 * (binInd2 - 1)) - 1]);
              if (t < 2.14748365E+9F) {
                if (t >= -2.14748365E+9F) {
                  i = static_cast<int32_T>(t);
                } else {
                  i = MIN_int32_T;
                }
              } else if (t >= 2.14748365E+9F) {
                i = MAX_int32_T;
              } else {
                i = 0;
              }
              if (distances_data[(static_cast<int32_T>(distInds[dInd]) +
                                  4 * (i - 1)) -
                                 1] < deltas[dInd]) {
                exitg1 = 1;
              } else if ((static_cast<int32_T>(
                             (static_cast<int32_T>(
                                 (!static_cast<int32_T>(
                                     pixelX < boundingBox_data[4 * (i - 1)])) &&
                                 (!static_cast<int32_T>(
                                     pixelX >
                                     boundingBox_data[4 * (i - 1) + 1])))) &&
                             (!static_cast<int32_T>(
                                 pixelY <
                                 boundingBox_data[4 * (i - 1) + 2])))) &&
                         (!static_cast<int32_T>(
                             pixelY > boundingBox_data[4 * (i - 1) + 3]))) {
                int32_T v;
                boolean_T exitg2;
                boolean_T y;
                v = 3 * (i - 1) + 1;
                r2 = v + 2;
                for (r1 = 0; r1 <= r2 - v; r1++) {
                  b_verts2d[r1 << 1] = allVerts2d_data[2 * ((v + r1) - 1)];
                  b_verts2d[(r1 << 1) + 1] =
                      allVerts2d_data[2 * ((v + r1) - 1) + 1];
                }
                point2d[0] = pixelX - b_verts2d[0];
                point2d[1] = pixelY - b_verts2d[1];
                for (r1 = 0; r1 < 3; r1++) {
                  verts2d[r1 << 1] = b_verts2d[r1 << 1] - b_verts2d[0];
                  verts2d[(r1 << 1) + 1] =
                      b_verts2d[(r1 << 1) + 1] - b_verts2d[1];
                }
                for (r1 = 0; r1 < 6; r1++) {
                  b_verts2d[r1] = verts2d[r1];
                }
                A[0] = b_verts2d[2];
                A[2] = b_verts2d[4];
                A[1] = b_verts2d[3];
                A[3] = b_verts2d[5];
                if (fabsf(b_verts2d[3]) > fabsf(b_verts2d[2])) {
                  r1 = 2;
                  r2 = 1;
                } else {
                  r1 = 1;
                  r2 = 2;
                }
                t = A[r2 - 1] / A[r1 - 1];
                u[1] = (point2d[r2 - 1] - point2d[r1 - 1] * t) /
                       (A[r2 + 1] - t * A[r1 + 1]);
                u[0] = (point2d[r1 - 1] - u[1] * A[r1 + 1]) / A[r1 - 1];
                x[0] = (u[0] < 0.0F);
                x[1] = (u[1] < 0.0F);
                y = false;
                r2 = 0;
                exitg2 = false;
                while ((!static_cast<int32_T>(exitg2)) &&
                       (static_cast<int32_T>(r2 < 2))) {
                  if (x[r2]) {
                    y = true;
                    exitg2 = true;
                  } else {
                    r2++;
                  }
                }
                if (!static_cast<int32_T>(y)) {
                  x[0] = (u[0] > 1.0F);
                  x[1] = (u[1] > 1.0F);
                  r2 = 0;
                  exitg2 = false;
                  while ((!static_cast<int32_T>(exitg2)) &&
                         (static_cast<int32_T>(r2 < 2))) {
                    if (x[r2]) {
                      y = true;
                      exitg2 = true;
                    } else {
                      r2++;
                    }
                  }
                  if ((!static_cast<int32_T>(y)) &&
                      (!static_cast<int32_T>(u[0] + u[1] > 1.0F))) {
                    real32_T d;
                    real32_T scale;
                    r2 = v + 2;
                    for (r1 = 0; r1 <= r2 - v; r1++) {
                      b_colors[3 * r1] =
                          allVertsCamera_data[3 * ((v + r1) - 1)];
                      b_colors[3 * r1 + 1] =
                          allVertsCamera_data[3 * ((v + r1) - 1) + 1];
                      b_colors[3 * r1 + 2] =
                          allVertsCamera_data[3 * ((v + r1) - 1) + 2];
                    }
                    for (r1 = 0; r1 < 2; r1++) {
                      colors[3 * r1] = b_colors[3 * (r1 + 1)] - b_colors[0];
                      colors[3 * r1 + 1] =
                          b_colors[3 * (r1 + 1) + 1] - b_colors[1];
                      colors[3 * r1 + 2] =
                          b_colors[3 * (r1 + 1) + 2] - b_colors[2];
                    }
                    d = 0.0F;
                    scale = 1.29246971E-26F;
                    for (r2 = 0; r2 < 3; r2++) {
                      absxk = fabsf(b_colors[r2] + (colors[r2] * u[0] +
                                                    colors[r2 + 3] * u[1]));
                      if (absxk > scale) {
                        t = scale / absxk;
                        d = d * t * t + 1.0F;
                        scale = absxk;
                      } else {
                        t = absxk / scale;
                        d += t * t;
                      }
                    }
                    d = scale * sqrtf(d);
                    if (d < depth) {
                      depth = d;
                      r2 = v + 2;
                      for (r1 = 0; r1 <= r2 - v; r1++) {
                        b_colors[3 * r1] = allColor_data[3 * ((v + r1) - 1)];
                        b_colors[3 * r1 + 1] =
                            allColor_data[3 * ((v + r1) - 1) + 1];
                        b_colors[3 * r1 + 2] =
                            allColor_data[3 * ((v + r1) - 1) + 2];
                      }
                      for (r1 = 0; r1 < 2; r1++) {
                        colors[3 * r1] = b_colors[3 * (r1 + 1)] - b_colors[0];
                        colors[3 * r1 + 1] =
                            b_colors[3 * (r1 + 1) + 1] - b_colors[1];
                        colors[3 * r1 + 2] =
                            b_colors[3 * (r1 + 1) + 2] - b_colors[2];
                      }
                      for (r1 = 0; r1 < 3; r1++) {
                        point[r1] = b_colors[r1] +
                                    (colors[r1] * u[0] + colors[r1 + 3] * u[1]);
                      }
                      faceInd = i;
                    }
                  } else {
                    //  miss
                  }
                } else {
                  //  miss
                }
              }
            } else {
              exitg1 = 1;
            }
          } while (exitg1 == 0);
        }
        t = roundf(tree_data[61 * (binInd2 - 1) + 4]);
        if (t < 2.14748365E+9F) {
          if (t >= -2.14748365E+9F) {
            binInd2 = static_cast<int32_T>(t);
          } else {
            binInd2 = MIN_int32_T;
          }
        } else if (t >= 2.14748365E+9F) {
          binInd2 = MAX_int32_T;
        } else {
          binInd2 = 0;
        }
      }
      t = roundf(tree_data[(childInd + 61 * (binInd - 1)) - 1]);
      if (t < 2.14748365E+9F) {
        if (t >= -2.14748365E+9F) {
          binInd = static_cast<int32_T>(t);
        } else {
          binInd = MIN_int32_T;
        }
      } else if (t >= 2.14748365E+9F) {
        binInd = MAX_int32_T;
      } else {
        binInd = 0;
      }
    }
    //  count
    im[((px << 9) - py) + 511] = point[0];
    im[((px << 9) - py) + 262655] = point[1];
    im[((px << 9) - py) + 524799] = point[2];
    vertInds[((px << 9) - py) + 511] = static_cast<real_T>(faceInd);
  }
}

void renderZBuffer(const real32_T allVerts2d_data[], const int32_T[2],
                   const real32_T allVertsCamera_data[], const int32_T[2],
                   const real32_T allColor_data[], const int32_T[2],
                   const real32_T boundingBox_data[], const int32_T[2],
                   const real32_T tree_data[], const int32_T[2],
                   const real32_T idxAll_data[], const int32_T idxAll_size[3],
                   const real32_T distances_data[], const int32_T[2],
                   real32_T im[786432], real_T vertInds[262144])
{
  int32_T(*gpu_idxAll_size)[3];
  hipMalloc(&gpu_idxAll_size, 12ULL);
  //  vertInds = coder.nullcopy(zeros(resX, resY, 'int32'));
  //  for px = int32(1):resX
  //      for py = int32(1):resY
  hipMemcpy(*gpu_idxAll_size, idxAll_size, 12ULL, hipMemcpyHostToDevice);
  renderZBuffer_kernel1<<<dim3(256U, 1U, 1U), dim3(1024U, 1U, 1U)>>>(
      allColor_data, allVertsCamera_data, allVerts2d_data, boundingBox_data,
      distances_data, idxAll_data, *gpu_idxAll_size, tree_data, vertInds, im);
  hipFree(*gpu_idxAll_size);
}

// End of code generation (renderZBuffer.cu)
