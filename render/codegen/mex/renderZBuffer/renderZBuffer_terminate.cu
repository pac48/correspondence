//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// renderZBuffer_terminate.cu
//
// Code generation for function 'renderZBuffer_terminate'
//

// Include files
#include "renderZBuffer_terminate.h"
#include "_coder_renderZBuffer_mex.h"
#include "renderZBuffer_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void renderZBuffer_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void renderZBuffer_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (renderZBuffer_terminate.cu)
