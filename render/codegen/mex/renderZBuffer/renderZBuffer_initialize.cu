//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// renderZBuffer_initialize.cu
//
// Code generation for function 'renderZBuffer_initialize'
//

// Include files
#include "renderZBuffer_initialize.h"
#include "_coder_renderZBuffer_mex.h"
#include "renderZBuffer_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void renderZBuffer_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (renderZBuffer_initialize.cu)
