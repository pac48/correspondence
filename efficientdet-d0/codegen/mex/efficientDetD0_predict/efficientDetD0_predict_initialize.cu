//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// efficientDetD0_predict_initialize.cu
//
// Code generation for function 'efficientDetD0_predict_initialize'
//

// Include files
#include "efficientDetD0_predict_initialize.h"
#include "_coder_efficientDetD0_predict_mex.h"
#include "efficientDetD0_predict.h"
#include "efficientDetD0_predict_data.h"
#include "rt_nonfinite.h"

// Function Declarations
static void efficientDetD0_predict_once();

// Function Definitions
static void efficientDetD0_predict_once()
{
  mex_InitInfAndNan();
  efficientDetD0_predict_init();
  hipMalloc(&xq_gpu_clone, sizeof(int8_T[8]));
  hipMalloc(&b_xq_gpu_clone, sizeof(int8_T[16]));
  hipMalloc(&c_xq_gpu_clone, sizeof(int8_T[32]));
  hipMalloc(&d_xq_gpu_clone, sizeof(int8_T[64]));
}

void efficientDetD0_predict_initialize()
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"neural_network_toolbox", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    efficientDetD0_predict_once();
  }
  hipGetLastError();
}

// End of code generation (efficientDetD0_predict_initialize.cu)
