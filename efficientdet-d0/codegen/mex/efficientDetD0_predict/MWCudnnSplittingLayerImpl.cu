#include "hip/hip_runtime.h"
#include "MWCudnnSplittingLayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include <cmath>
#include <cassert>
 namespace MWCudnnTarget { 
MWSplittingLayerImpl::MWSplittingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int numOutputs, int* channelSizes) : 
MWCNNLayerImpl(layer, ntwk_impl) , etjQLJVQCaeAXRWYtqOl(numOutputs) , 
MgAiRWiTutoTMxKXjmHQ(channelSizes) { } 
MWSplittingLayerImpl::~MWSplittingLayerImpl() { } void 
MWSplittingLayerImpl::propagateSize() { } void __global__ 
__launch_bounds__(1024) SplittingLayerImpl(float* inputBuffer, float* 
outputBuffer, int MW_startingChannel, int MW_channelWidth,  double 
YNmJhGSUszJKxsodxiuV, double YNDVziqpDddiXQKYZZhX, double 
YMNbgnUYZspjMLjwcIOS, double hljcfGWsvZXJZNrImpJB, const long 
int BRSPqxNffoBYKqpSVHne) { int idx = blockDim.x * blockIdx.x + threadIdx.x; if (idx < 
BRSPqxNffoBYKqpSVHne) { double cAUupmktEnGPfLHyWfFm = floor(idx / 
YMNbgnUYZspjMLjwcIOS); double EpwuhXsRcwdqXSjBpUeO = idx - 
(YMNbgnUYZspjMLjwcIOS * cAUupmktEnGPfLHyWfFm); double LklYEpYUjaLTgcFFAaJX = 
floor(EpwuhXsRcwdqXSjBpUeO / YNDVziqpDddiXQKYZZhX); if 
(static_cast<int>(LklYEpYUjaLTgcFFAaJX) >= MW_startingChannel && 
static_cast<int>(LklYEpYUjaLTgcFFAaJX) < MW_startingChannel + MW_channelWidth ) { 
double LtEgcYoEYjkrWuohutgw = LklYEpYUjaLTgcFFAaJX - (MW_startingChannel); double 
EvebzoroiuKkIxwjkGnD = EpwuhXsRcwdqXSjBpUeO - 
(YNDVziqpDddiXQKYZZhX * LklYEpYUjaLTgcFFAaJX); long int opIdx = cAUupmktEnGPfLHyWfFm 
* hljcfGWsvZXJZNrImpJB + LtEgcYoEYjkrWuohutgw * 
YNDVziqpDddiXQKYZZhX + EvebzoroiuKkIxwjkGnD; outputBuffer[opIdx] 
= inputBuffer[idx]; } } } void MWSplittingLayerImpl::predict() { MWTensorBase* 
ipTensorBase = getLayer()->getInputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); if (getLayer()->getInPlaceIndex(0) 
!= -1) { assert(ipTensor->getBatchSize() == 1); 
assert(ipTensor->getSequenceLength() == 1); } else { int YNmJhGSUszJKxsodxiuV = 
ipTensor->getWidth(); long int YNDVziqpDddiXQKYZZhX = 
ipTensor->getHeight() * ipTensor->getWidth(); long int 
YMNbgnUYZspjMLjwcIOS = YNDVziqpDddiXQKYZZhX * 
(ipTensor->getChannels()); long int YGiQICncmsGZkNUyiQyg = 
YMNbgnUYZspjMLjwcIOS * ipTensor->getBatchSize(); long int 
rlQsibXJSWJVnUVpdNeL = ((YGiQICncmsGZkNUyiQyg + 31) / 32) * 32; long int 
shEncNmxJsMuJKwbrwok = (rlQsibXJSWJVnUVpdNeL < 1024) ? rlQsibXJSWJVnUVpdNeL : 1024; 
long int KHClOltUSuqFVVErSxVb = (YGiQICncmsGZkNUyiQyg + 
shEncNmxJsMuJKwbrwok - 1) / shEncNmxJsMuJKwbrwok; int 
MW_startingChannel = 0; for (int i = 0; i < this->etjQLJVQCaeAXRWYtqOl; i++){  
long int hljcfGWsvZXJZNrImpJB = YNDVziqpDddiXQKYZZhX * 
(getLayer()->getOutputTensor(i)->getChannels()); 
SplittingLayerImpl<<<KHClOltUSuqFVVErSxVb, shEncNmxJsMuJKwbrwok>>>( 
ipTensor->getData(), 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(i))->getData(), 
MW_startingChannel, getLayer()->getOutputTensor(i)->getChannels(), 
YNmJhGSUszJKxsodxiuV, YNDVziqpDddiXQKYZZhX, 
YMNbgnUYZspjMLjwcIOS, hljcfGWsvZXJZNrImpJB, 
YGiQICncmsGZkNUyiQyg); MW_startingChannel += 
getLayer()->getOutputTensor(i)->getChannels(); } } return; } void 
MWSplittingLayerImpl::cleanup() { } float* 
MWSplittingLayerImpl::offsetOutputPointer(float* initialPointer, int outIdx) { 
if (getLayer()->getInPlaceIndex(0) != -1) { 
assert(getLayer()->getInputTensor(0)->getBatchSize() == 1); 
assert(getLayer()->getInputTensor(0)->getSequenceLength() == 1);  long int 
accumulatedC = 0; long int HW = (getLayer()->getOutputTensor(0)->getHeight()) * 
(getLayer()->getOutputTensor(0)->getWidth()); assert(outIdx < 
this->etjQLJVQCaeAXRWYtqOl); for (int i = 0; i < outIdx; i++){ accumulatedC += 
getLayer()->getOutputTensor(i)->getChannels(); } return initialPointer + 
(HW*accumulatedC); } else { return initialPointer; } } signed char* 
MWSplittingLayerImpl::offsetOutputPointer(signed char* initialPointer, int 
outIdx) { assert(false); return NULL; } } 