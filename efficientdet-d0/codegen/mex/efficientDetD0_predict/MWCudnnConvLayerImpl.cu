#include "MWCudnnConvLayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include <cassert>
#include <cstdio>
#include <vector>
 namespace MWCudnnTarget { MWConvLayerImpl::MWConvLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int filt_H, int filt_W, int numGrps, int 
numChnls, int numFilts, int FOcStuqCptsGIZXskVpC, int 
FpguQZSermqZCMRiUfML, int CTCbzQMDaLxINPbODdng, int 
CLOUhPjbgggWoXHTtmjC, int CpMjJjtGOeWOzwxpAAQP, int 
CqtPRJvHlGJFssiPzsOm, int AjhVZuQXURJimwbnYqDF, int 
AwZQzUhuWVLGrWgLHRuM, const char* xHViLEwTujGGrPZZgmbF, const 
char* JxwPQNPACGfmGpNncpCY) : MWCNNLayerImpl(layer, ntwk_impl) , 
vIWQzNvYZSuxmOTVDFhU(NULL) , IwKnaBoXVubIRYcxEJLH(NULL) , SugesRlPIbOVzRgNWRnl(NULL) , 
AzTsxYcYjIEJsGQbeYHm(filt_H) , BHuHNDGoRwGRouCxeMbw (filt_W) , 
BkwhtPQUCQKchmmimoXs (numGrps) , BUOdotSvmFyUWQKMUdra (numChnls) , 
BdqURaHPmdnfzvtUvocl (numFilts) , 
AdmgfUbRAfzFeYHxSnQr(AjhVZuQXURJimwbnYqDF) , 
AuqaQHxmPQSyYRemQvyX(AwZQzUhuWVLGrWgLHRuM) , 
CGbFsczkgkhjcHoCKzBx(CTCbzQMDaLxINPbODdng) , 
CDJtexcMbXMWAmnNZsNf(CLOUhPjbgggWoXHTtmjC) , 
CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP) , 
ClEhcJFlvGCgiavziIag(CqtPRJvHlGJFssiPzsOm) , 
FLuSVNoPhAFKtLUchSvv(FOcStuqCptsGIZXskVpC) , 
FeVcBgtQmTLtmnNcJGMY(FpguQZSermqZCMRiUfML) , 
IAlDgIFcchbwRGBSfVfA((CGbFsczkgkhjcHoCKzBx != CDJtexcMbXMWAmnNZsNf) 
|| (CZNYmBcNFSZWvaCklqeM != ClEhcJFlvGCgiavziIag)) { cQBKlCKXxecGPJrXBXdk = 
ntwk_impl; CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&NMMfJylfQjiIUAKhXCJb)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&PtkeOkuClHzhOfpmBevf)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JsZenQeBPMhwsyEhVHiD));  MWCNNLayer* 
convLayer = getLayer(); MWTensorBase* ipTensor = convLayer->getInputTensor(0); 
if (IAlDgIFcchbwRGBSfVfA) { SugesRlPIbOVzRgNWRnl = new MWTensor<float>(-1, 
-1, -1, -1, -1, NULL, getLayer(), ipTensor->getDataFormat(), 0); if 
(!SugesRlPIbOVzRgNWRnl) { MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&XYbzSmRQGatVJtGmDZSo)); } else { 
SugesRlPIbOVzRgNWRnl = ipTensor; } assert(SugesRlPIbOVzRgNWRnl != NULL); int 
NNhshzQGJHLSGjDiVerE; int NXruhrCCiguRjAgSNDuz; if 
(IAlDgIFcchbwRGBSfVfA) { NNhshzQGJHLSGjDiVerE = 0;  
NXruhrCCiguRjAgSNDuz = 0; } else { NNhshzQGJHLSGjDiVerE = 
CGbFsczkgkhjcHoCKzBx; NXruhrCCiguRjAgSNDuz = CZNYmBcNFSZWvaCklqeM; } 
bERCRkGjpaKXMNComoYl = CGbFsczkgkhjcHoCKzBx; bOrQjJTNlssnrexxbHdi = CZNYmBcNFSZWvaCklqeM;
#if (CUDNN_MAJOR <= 5)
 { if ((AdmgfUbRAfzFeYHxSnQr != 1) && (AuqaQHxmPQSyYRemQvyX != 1)){ 
printf("Dilated Convolution only supported for cuDNN 6 or greater "); throw 
std::runtime_error("Unsupported Dilation Factor"); } 
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(NMMfJylfQjiIUAKhXCJb, 
NNhshzQGJHLSGjDiVerE, NXruhrCCiguRjAgSNDuz, FLuSVNoPhAFKtLUchSvv, 
FeVcBgtQmTLtmnNcJGMY, 1, 1, HIPDNN_CROSS_CORRELATION));  }
#else
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(NMMfJylfQjiIUAKhXCJb, 
NNhshzQGJHLSGjDiVerE, NXruhrCCiguRjAgSNDuz, FLuSVNoPhAFKtLUchSvv, 
FeVcBgtQmTLtmnNcJGMY, AdmgfUbRAfzFeYHxSnQr, AuqaQHxmPQSyYRemQvyX, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT)); }
#endif
#if (FP16_ENABLED == 1 && ( CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2) ))
 CUDNN_CALL(hipdnnSetConvolutionMathType(NMMfJylfQjiIUAKhXCJb, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
 if (BkwhtPQUCQKchmmimoXs > 1){ 
CUDNN_CALL(hipdnnSetConvolutionGroupCount(NMMfJylfQjiIUAKhXCJb, 
BkwhtPQUCQKchmmimoXs)); } int eWYFXrUazhqiEIscccda = 
BUOdotSvmFyUWQKMUdra*BkwhtPQUCQKchmmimoXs; int eqmVWbEcwBRGnVNDUtrG = 
BdqURaHPmdnfzvtUvocl*BkwhtPQUCQKchmmimoXs; 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(PtkeOkuClHzhOfpmBevf, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, eqmVWbEcwBRGnVNDUtrG, 
eWYFXrUazhqiEIscccda/BkwhtPQUCQKchmmimoXs, AzTsxYcYjIEJsGQbeYHm, 
BHuHNDGoRwGRouCxeMbw)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JsZenQeBPMhwsyEhVHiD, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, eqmVWbEcwBRGnVNDUtrG, 1, 1)); int weightSize = 
BUOdotSvmFyUWQKMUdra*eqmVWbEcwBRGnVNDUtrG*AzTsxYcYjIEJsGQbeYHm*BHuHNDGoRwGRouCxeMbw; 
CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, sizeof(float)*weightSize)); 
CUDA_CALL(hipMalloc((void**)&IwKnaBoXVubIRYcxEJLH, 
sizeof(float)*eqmVWbEcwBRGnVNDUtrG)); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(JxwPQNPACGfmGpNncpCY); 
createAndAddDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); } 
void MWConvLayerImpl::propagateSize() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); int inputH; int inputW; if 
(IAlDgIFcchbwRGBSfVfA) { inputH = ipTensor->getHeight() + 
CGbFsczkgkhjcHoCKzBx + CDJtexcMbXMWAmnNZsNf; inputW = ipTensor->getWidth() + 
CZNYmBcNFSZWvaCklqeM + ClEhcJFlvGCgiavziIag; } else { inputH = 
ipTensor->getHeight(); inputW = ipTensor->getWidth(); } 
SugesRlPIbOVzRgNWRnl->setHeight(inputH); SugesRlPIbOVzRgNWRnl->setWidth(inputW); 
SugesRlPIbOVzRgNWRnl->setChannels(ipTensor->getChannels()); 
SugesRlPIbOVzRgNWRnl->setBatchSize(ipTensor->getBatchSize()); 
SugesRlPIbOVzRgNWRnl->setSequenceLength(ipTensor->getSequenceLength()); 
assert(SugesRlPIbOVzRgNWRnl->getSequenceLength() == 1); if 
(IAlDgIFcchbwRGBSfVfA) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(XYbzSmRQGatVJtGmDZSo, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, SugesRlPIbOVzRgNWRnl->getBatchSize(), SugesRlPIbOVzRgNWRnl->getChannels(), 
SugesRlPIbOVzRgNWRnl->getHeight(), SugesRlPIbOVzRgNWRnl->getWidth())); } else { 
XYbzSmRQGatVJtGmDZSo = MWCNNLayerImpl::getCuDNNDescriptor(SugesRlPIbOVzRgNWRnl); } 
MWTensorBase* opTensor = getLayer()->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));
#if (CUDNN_MAJOR < 7)
 { 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, PtkeOkuClHzhOfpmBevf, NMMfJylfQjiIUAKhXCJb, *desc, 
HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NDjzAZSYJuWymuKDNZYB)); }
#else
 { const int maxAlgoCount(3); int returnedAlgoCount(-1);  
hipdnnConvolutionFwdAlgoPerf_t perf_results[maxAlgoCount]; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, PtkeOkuClHzhOfpmBevf, NMMfJylfQjiIUAKhXCJb, *desc, 
maxAlgoCount, &returnedAlgoCount, perf_results)); NDjzAZSYJuWymuKDNZYB = 
perf_results[0].algo; }
#endif
 size_t sxuOMwKXOKfuExclRaSe = 0; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize( 
*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
PtkeOkuClHzhOfpmBevf, NMMfJylfQjiIUAKhXCJb, *desc, NDjzAZSYJuWymuKDNZYB, 
&sxuOMwKXOKfuExclRaSe)); if (sxuOMwKXOKfuExclRaSe > 
*cQBKlCKXxecGPJrXBXdk->getProposedWorkSpaceSize()) { 
cQBKlCKXxecGPJrXBXdk->setProposedWorkSpaceSize(sxuOMwKXOKfuExclRaSe); } } void 
MWConvLayerImpl::allocate() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); if (IAlDgIFcchbwRGBSfVfA) { float* 
newInput; int inputH = ipTensor->getHeight() + CGbFsczkgkhjcHoCKzBx + 
CDJtexcMbXMWAmnNZsNf; int inputW = ipTensor->getWidth() + 
CZNYmBcNFSZWvaCklqeM + ClEhcJFlvGCgiavziIag; int paddedSize = 
ipTensor->getBatchSize() * ipTensor->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float)*paddedSize)); 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->setData(newInput); } } void 
MWConvLayerImpl::deallocate() { if (SugesRlPIbOVzRgNWRnl != 
getLayer()->getInputTensor(0)) { assert(IAlDgIFcchbwRGBSfVfA); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData()); 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->setData((float*)NULL); } } void 
MWConvLayerImpl::predict() { MWCNNLayer* convLayer = getLayer(); MWTensorBase* 
ipTensorBase = convLayer->getInputTensor(); MWTensorBase* opTensorBase = 
convLayer->getOutputTensor(); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); if (SugesRlPIbOVzRgNWRnl != 
convLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), 
0, sizeof(float)*SugesRlPIbOVzRgNWRnl->getNumElements()));  
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), SugesRlPIbOVzRgNWRnl->getHeight(), 
SugesRlPIbOVzRgNWRnl->getWidth(), bERCRkGjpaKXMNComoYl, bOrQjJTNlssnrexxbHdi, 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), 
ipTensor->getNumElements()); } assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData()); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); 
CUDNN_CALL(hipdnnConvolutionForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
getOnePtr(), XYbzSmRQGatVJtGmDZSo, 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), PtkeOkuClHzhOfpmBevf, 
vIWQzNvYZSuxmOTVDFhU, NMMfJylfQjiIUAKhXCJb, NDjzAZSYJuWymuKDNZYB, 
cQBKlCKXxecGPJrXBXdk->getWorkSpace(), 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize(), getZeroPtr(), *desc, 
opTensor->getData())); 
CUDNN_CALL(hipdnnAddTensor(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), getOnePtr(), 
JsZenQeBPMhwsyEhVHiD, IwKnaBoXVubIRYcxEJLH, getOnePtr(), *desc, opTensor->getData())); } 
void MWConvLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(NMMfJylfQjiIUAKhXCJb)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(PtkeOkuClHzhOfpmBevf)); if 
(vIWQzNvYZSuxmOTVDFhU) { CUDA_FREE_CALL(vIWQzNvYZSuxmOTVDFhU); vIWQzNvYZSuxmOTVDFhU = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JsZenQeBPMhwsyEhVHiD)); if 
(IwKnaBoXVubIRYcxEJLH) { CUDA_FREE_CALL(IwKnaBoXVubIRYcxEJLH); IwKnaBoXVubIRYcxEJLH = NULL; } if 
(SugesRlPIbOVzRgNWRnl != getLayer()->getInputTensor(0)) { 
assert(IAlDgIFcchbwRGBSfVfA); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(XYbzSmRQGatVJtGmDZSo)); } } void 
MWConvLayerImpl::loadWeights(const char* PmFfARVzoHVAYkfpuvqK) {  FILE* 
QMgBqCuvjnbWHWiVPEwn = MWCNNLayer::openBinaryFile(PmFfARVzoHVAYkfpuvqK); 
assert(QMgBqCuvjnbWHWiVPEwn); int cRtIUoZRPICuQEOZOSzT = 
BUOdotSvmFyUWQKMUdra*BdqURaHPmdnfzvtUvocl*BkwhtPQUCQKchmmimoXs*AzTsxYcYjIEJsGQbeYHm*BHuHNDGoRwGRouCxeMbw; 
 float* KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float)*cRtIUoZRPICuQEOZOSzT); 
MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), cRtIUoZRPICuQEOZOSzT, 
QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK); CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, 
KZWeXiYFmdpQdsgidKeG, sizeof(float)*cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); 
fclose(QMgBqCuvjnbWHWiVPEwn); free(KZWeXiYFmdpQdsgidKeG); } void 
MWConvLayerImpl::loadBias(const char* PmFfARVzoHVAYkfpuvqK) { FILE* 
QMgBqCuvjnbWHWiVPEwn = MWCNNLayer::openBinaryFile(PmFfARVzoHVAYkfpuvqK);  
assert(QMgBqCuvjnbWHWiVPEwn); int cRtIUoZRPICuQEOZOSzT = 
BdqURaHPmdnfzvtUvocl*BkwhtPQUCQKchmmimoXs;  float* KZWeXiYFmdpQdsgidKeG = 
MALLOC_CALL(sizeof(float)*cRtIUoZRPICuQEOZOSzT); 
MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), cRtIUoZRPICuQEOZOSzT, 
QMgBqCuvjnbWHWiVPEwn, PmFfARVzoHVAYkfpuvqK); CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, 
KZWeXiYFmdpQdsgidKeG, sizeof(float)*cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QMgBqCuvjnbWHWiVPEwn); } void 
MWConvLayerImpl::setLearnables(std::vector<float*> learnables) { 
assert(learnables.size() == 2); int cRtIUoZRPICuQEOZOSzT = BUOdotSvmFyUWQKMUdra * 
BdqURaHPmdnfzvtUvocl * BkwhtPQUCQKchmmimoXs * AzTsxYcYjIEJsGQbeYHm * 
BHuHNDGoRwGRouCxeMbw;  float* wqggPBXZvtlxnxwngvAq = learnables[0]; 
CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, wqggPBXZvtlxnxwngvAq, sizeof(float) * 
cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); cRtIUoZRPICuQEOZOSzT = 
BdqURaHPmdnfzvtUvocl * BkwhtPQUCQKchmmimoXs;  float* JgLfgHrHMEMmMYTettJF = 
learnables[1]; CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, JgLfgHrHMEMmMYTettJF, 
sizeof(float) * cRtIUoZRPICuQEOZOSzT, hipMemcpyHostToDevice)); } void 
MWConvLayerImpl::postSetup() { if (cQBKlCKXxecGPJrXBXdk->getAutoTune()) { 
getConvAlgoTuned(); } else { getConvAlgoWorkSpaceLimit(); } } void 
MWConvLayerImpl::getConvAlgoTuned() { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); 
assert(desc); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), 
PtkeOkuClHzhOfpmBevf, vIWQzNvYZSuxmOTVDFhU, NMMfJylfQjiIUAKhXCJb, *desc, 
opTensor->getData(), 3, &returnedAlgoCount, &perf_results[0], 
cQBKlCKXxecGPJrXBXdk->getWorkSpace(), 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize())); NDjzAZSYJuWymuKDNZYB = 
perf_results[0].algo; } void MWConvLayerImpl::getConvAlgoWorkSpaceLimit() { 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR < 8)
 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, PtkeOkuClHzhOfpmBevf, NMMfJylfQjiIUAKhXCJb, *desc, 
HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, 
*cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize(), &NDjzAZSYJuWymuKDNZYB));
#else
 int maxAlgoCount(-1); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
&maxAlgoCount)); int returnedAlgoCount(-1); 
std::vector<hipdnnConvolutionFwdAlgoPerf_t> perf_results(maxAlgoCount);  
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, PtkeOkuClHzhOfpmBevf, NMMfJylfQjiIUAKhXCJb, *desc, 
maxAlgoCount, &returnedAlgoCount, &perf_results[0])); 
hipdnnConvolutionFwdAlgoPerf_t nextFastest; bool algoFound(false); for (int i = 
0; i < returnedAlgoCount; ++i) { nextFastest = perf_results[i]; if 
(nextFastest.memory <= *cQBKlCKXxecGPJrXBXdk->getAllocatedWorkSpaceSize()) { 
NDjzAZSYJuWymuKDNZYB = nextFastest.algo; algoFound = true; break; } } assert(algoFound);
#endif
 } } 