#include "hip/hip_runtime.h"
#include "MWCudnnAvgPoolingLayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
 namespace MWCudnnTarget { 
MWAvgPoolingLayerImpl::MWAvgPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int DGzdAcREJHGXjyRzNjJV, int 
ECTnqgWHyHCHCLBZlffd, int FOcStuqCptsGIZXskVpC, int FpguQZSermqZCMRiUfML, 
int CTCbzQMDaLxINPbODdng, int CLOUhPjbgggWoXHTtmjC, int 
CpMjJjtGOeWOzwxpAAQP, int CqtPRJvHlGJFssiPzsOm) : 
MWCNNLayerImpl(layer, ntwk_impl) , SugesRlPIbOVzRgNWRnl(NULL) , 
DCdZnqpcBnvXVgEsLBnz(DGzdAcREJHGXjyRzNjJV) , 
DqxLTLaJwwgQqmrtCDuu(ECTnqgWHyHCHCLBZlffd) , 
CufLFODQDXTAPyRqYodN(DGzdAcREJHGXjyRzNjJV) , 
DSsxcjIrUgZCKZovyNQf(ECTnqgWHyHCHCLBZlffd) , 
FLuSVNoPhAFKtLUchSvv(FOcStuqCptsGIZXskVpC) , 
FeVcBgtQmTLtmnNcJGMY(FpguQZSermqZCMRiUfML) , 
CGbFsczkgkhjcHoCKzBx(CTCbzQMDaLxINPbODdng) , 
CDJtexcMbXMWAmnNZsNf(CLOUhPjbgggWoXHTtmjC) , 
CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP) , 
ClEhcJFlvGCgiavziIag(CqtPRJvHlGJFssiPzsOm) , 
IAlDgIFcchbwRGBSfVfA((CGbFsczkgkhjcHoCKzBx != CDJtexcMbXMWAmnNZsNf) 
|| (CZNYmBcNFSZWvaCklqeM != ClEhcJFlvGCgiavziIag)) , 
lWJYwWaFPmWNQDPrlqER(CTCbzQMDaLxINPbODdng) , 
lXJKIOEATumoVKStGbVy(CpMjJjtGOeWOzwxpAAQP) , 
puSFZkRJmyuFPfQRswDK(std::pow(2, layer->getScalingExponent())) , 
vFNECEAeLZsYsUxvlgqL(std::string{"INT8x4"}.compare(layer->getAccelMode()) 
== 0){ CUDNN_CALL(hipdnnCreatePoolingDescriptor(&lHtftnmGBvlSSoGOXVui)); 
MWTensorBase* ipTensor = getLayer()->getInputTensor(0); if 
(IAlDgIFcchbwRGBSfVfA) { assert(ipTensor->isFloat()); 
lWJYwWaFPmWNQDPrlqER = 0;  lXJKIOEATumoVKStGbVy = 0; 
SugesRlPIbOVzRgNWRnl = new MWTensor<float>(-1, -1, -1, -1, -1, NULL, getLayer(), 
ipTensor->getDataFormat(), 0); if (!SugesRlPIbOVzRgNWRnl) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&XYbzSmRQGatVJtGmDZSo));  } else { 
SugesRlPIbOVzRgNWRnl = ipTensor;  } assert(SugesRlPIbOVzRgNWRnl != NULL); MWTensorBase* 
opTensorBase = getLayer()->getOutputTensor(0); 
createAndAddDescriptor(opTensorBase->getSourcePortIndex()); if 
(opTensorBase->isInt8()) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&IpFhwalnAlrMvcuyQpQD)); } } 
MWAvgPoolingLayerImpl::~MWAvgPoolingLayerImpl() { } void 
MWAvgPoolingLayerImpl::propagateSize() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0);  if ((DCdZnqpcBnvXVgEsLBnz == -1) && 
(DqxLTLaJwwgQqmrtCDuu == -1)) { CufLFODQDXTAPyRqYodN = 
ipTensorBase->getHeight(); DSsxcjIrUgZCKZovyNQf = ipTensorBase->getWidth(); } int 
inputH; int inputW; if (IAlDgIFcchbwRGBSfVfA) { inputH = 
ipTensorBase->getHeight() + CGbFsczkgkhjcHoCKzBx + CDJtexcMbXMWAmnNZsNf;  
inputW = ipTensorBase->getWidth() + CZNYmBcNFSZWvaCklqeM + 
ClEhcJFlvGCgiavziIag; } else { inputH = ipTensorBase->getHeight(); inputW = 
ipTensorBase->getWidth(); } SugesRlPIbOVzRgNWRnl->setHeight(inputH); 
SugesRlPIbOVzRgNWRnl->setWidth(inputW); 
SugesRlPIbOVzRgNWRnl->setChannels(ipTensorBase->getChannels()); 
SugesRlPIbOVzRgNWRnl->setBatchSize(ipTensorBase->getBatchSize()); 
SugesRlPIbOVzRgNWRnl->setSequenceLength(ipTensorBase->getSequenceLength()); 
assert(SugesRlPIbOVzRgNWRnl->getSequenceLength() == 1); 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(lHtftnmGBvlSSoGOXVui,  
HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,  HIPDNN_NOT_PROPAGATE_NAN,  
CufLFODQDXTAPyRqYodN,  DSsxcjIrUgZCKZovyNQf,  lWJYwWaFPmWNQDPrlqER,  
lXJKIOEATumoVKStGbVy,  FLuSVNoPhAFKtLUchSvv,  FeVcBgtQmTLtmnNcJGMY));  
hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); if 
(opTensorBase->isInt8()) { if (vFNECEAeLZsYsUxvlgqL) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(IpFhwalnAlrMvcuyQpQD, 
HIPDNN_TENSOR_NCHW_VECT_C,  HIPDNN_DATA_INT8x4,  ipTensorBase->getBatchSize(),  
ipTensorBase->getChannels(),  ipTensorBase->getHeight(),  
ipTensorBase->getWidth())  ); MWCNNLayerImpl::setDescriptorForINT8(*desc, 
static_cast<MWTensor<signed char>*>(opTensorBase), HIPDNN_DATA_INT8x4, 
HIPDNN_TENSOR_NCHW_VECT_C );  } else { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(IpFhwalnAlrMvcuyQpQD, 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_INT8, ipTensorBase->getBatchSize(), 
ipTensorBase->getChannels(), ipTensorBase->getHeight(), 
ipTensorBase->getWidth())); MWCNNLayerImpl::setDescriptorForINT8(*desc, 
static_cast<MWTensor<signed char>*>(opTensorBase), HIPDNN_DATA_INT8, 
HIPDNN_TENSOR_NCHW); } } else { if (IAlDgIFcchbwRGBSfVfA) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(XYbzSmRQGatVJtGmDZSo, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, SugesRlPIbOVzRgNWRnl->getBatchSize(), SugesRlPIbOVzRgNWRnl->getChannels(), 
SugesRlPIbOVzRgNWRnl->getHeight(), SugesRlPIbOVzRgNWRnl->getWidth())); } else { 
XYbzSmRQGatVJtGmDZSo = MWCNNLayerImpl::getCuDNNDescriptor(SugesRlPIbOVzRgNWRnl); }  
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensorBase)); } } 
void MWAvgPoolingLayerImpl::allocate() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); if (IAlDgIFcchbwRGBSfVfA) { float* 
newInput; int inputH = ipTensorBase->getHeight() + CGbFsczkgkhjcHoCKzBx + 
CDJtexcMbXMWAmnNZsNf;  int inputW = ipTensorBase->getWidth() + 
CZNYmBcNFSZWvaCklqeM + ClEhcJFlvGCgiavziIag; int paddedSize = 
ipTensorBase->getBatchSize() * ipTensorBase->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float)*paddedSize)); 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->setData(newInput); } } void 
MWAvgPoolingLayerImpl::deallocate() { if (SugesRlPIbOVzRgNWRnl != 
getLayer()->getInputTensor(0)) { assert(IAlDgIFcchbwRGBSfVfA); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData()); 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->setData((float*)NULL); } } void 
MWAvgPoolingLayerImpl::predict() { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); if 
(opTensorBase->isInt8()) { assert(!IAlDgIFcchbwRGBSfVfA); 
MWTensor<signed char>* opTensor = static_cast<MWTensor<signed 
char>*>(opTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
lHtftnmGBvlSSoGOXVui, &puSFZkRJmyuFPfQRswDK, 
IpFhwalnAlrMvcuyQpQD, static_cast<MWTensor<signed 
char>*>(SugesRlPIbOVzRgNWRnl)->getData(), getZeroPtr(), *desc, opTensor->getData())); 
}else{ MWTensor<float>* ipTensor = static_cast<MWTensor<float>*>(ipTensorBase); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); if 
(SugesRlPIbOVzRgNWRnl != ipTensorBase) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), 
0, sizeof(float)*SugesRlPIbOVzRgNWRnl->getNumElements())); 
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), SugesRlPIbOVzRgNWRnl->getHeight(), 
SugesRlPIbOVzRgNWRnl->getWidth(), CGbFsczkgkhjcHoCKzBx, CZNYmBcNFSZWvaCklqeM, 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), 
ipTensor->getNumElements()); } assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData()); 
CUDNN_CALL(hipdnnPoolingForward(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), 
lHtftnmGBvlSSoGOXVui, getOnePtr(), XYbzSmRQGatVJtGmDZSo, 
static_cast<MWTensor<float>*>(SugesRlPIbOVzRgNWRnl)->getData(), getZeroPtr(), *desc, 
opTensor->getData())); } } void MWAvgPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(lHtftnmGBvlSSoGOXVui)); if 
(SugesRlPIbOVzRgNWRnl != getLayer()->getInputTensor(0)) { 
assert(IAlDgIFcchbwRGBSfVfA); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(XYbzSmRQGatVJtGmDZSo)); } MWTensorBase* 
opTensorBase = getLayer()->getOutputTensor(0); if (opTensorBase->isInt8()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(IpFhwalnAlrMvcuyQpQD)); }  } } 