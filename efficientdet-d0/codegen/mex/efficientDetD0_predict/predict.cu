#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// predict.cu
//
// Code generation for function 'predict'
//

// Include files
#include "predict.h"
#include "dlnetwork.h"
#include "efficientDetD0_predict_data.h"
#include "efficientDetD0_predict_internal_types.h"
#include "efficientDetD0_predict_mexutil.h"
#include "efficientDetD0_predict_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"

// Type Definitions
struct cell_wrap_6 {
  real32_T f1[786432];
};

// Function Declarations
static __global__ void
dlnetwork_predict_kernel1(const coder::k_dlarray *varargin_1,
                          cell_wrap_6 inputDataT[1]);

static __global__ void
dlnetwork_predict_kernel10(const real32_T output[3317760],
                           real32_T varargout_9_Data[3317760]);

static __global__ void
dlnetwork_predict_kernel11(const real32_T output[147456],
                           real32_T varargout_10_Data[147456]);

static __global__ void
dlnetwork_predict_kernel2(const real32_T output[12960],
                          real32_T varargout_1_Data[12960]);

static __global__ void
dlnetwork_predict_kernel3(const real32_T output[576],
                          real32_T varargout_2_Data[576]);

static __global__ void
dlnetwork_predict_kernel4(const real32_T output[51840],
                          real32_T varargout_3_Data[51840]);

static __global__ void
dlnetwork_predict_kernel5(const real32_T output[2304],
                          real32_T varargout_4_Data[2304]);

static __global__ void
dlnetwork_predict_kernel6(const real32_T output[207360],
                          real32_T varargout_5_Data[207360]);

static __global__ void
dlnetwork_predict_kernel7(const real32_T output[9216],
                          real32_T varargout_6_Data[9216]);

static __global__ void
dlnetwork_predict_kernel8(const real32_T output[829440],
                          real32_T varargout_7_Data[829440]);

static __global__ void
dlnetwork_predict_kernel9(const real32_T output[36864],
                          real32_T varargout_8_Data[36864]);

// Function Definitions
static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel1(
    const coder::k_dlarray *varargin_1, cell_wrap_6 inputDataT[1])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 512ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 512ULL;
  i1 = static_cast<int32_T>(threadId % 512ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 512ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 3)) &&
                            (static_cast<int32_T>(i1 < 512)))) &&
      (static_cast<int32_T>(i < 512))) {
    inputDataT[0].f1[(i + (i1 << 9)) + (p << 18)] =
        varargin_1->Data[(i1 + (i << 9)) + (p << 18)];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel10(
    const real32_T output[3317760], real32_T varargout_9_Data[3317760])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 64ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 64ULL;
  i1 = static_cast<int32_T>(threadId % 64ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 64ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 810)) &&
                            (static_cast<int32_T>(i1 < 64)))) &&
      (static_cast<int32_T>(i < 64))) {
    varargout_9_Data[(i + (i1 << 6)) + (p << 12)] =
        output[(i1 + (i << 6)) + (p << 12)];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel11(
    const real32_T output[147456], real32_T varargout_10_Data[147456])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 64ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 64ULL;
  i1 = static_cast<int32_T>(threadId % 64ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 64ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 36)) &&
                            (static_cast<int32_T>(i1 < 64)))) &&
      (static_cast<int32_T>(i < 64))) {
    varargout_10_Data[(i + (i1 << 6)) + (p << 12)] =
        output[(i1 + (i << 6)) + (p << 12)];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel2(
    const real32_T output[12960], real32_T varargout_1_Data[12960])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 4ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 4ULL;
  i1 = static_cast<int32_T>(threadId % 4ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 4ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 810)) &&
                            (static_cast<int32_T>(i1 < 4)))) &&
      (static_cast<int32_T>(i < 4))) {
    varargout_1_Data[(i + (i1 << 2)) + (p << 4)] =
        output[(i1 + (i << 2)) + (p << 4)];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel3(
    const real32_T output[576], real32_T varargout_2_Data[576])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 4ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 4ULL;
  i1 = static_cast<int32_T>(threadId % 4ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 4ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 36)) &&
                            (static_cast<int32_T>(i1 < 4)))) &&
      (static_cast<int32_T>(i < 4))) {
    varargout_2_Data[(i + (i1 << 2)) + (p << 4)] =
        output[(i1 + (i << 2)) + (p << 4)];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel4(
    const real32_T output[51840], real32_T varargout_3_Data[51840])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 8ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 8ULL;
  i1 = static_cast<int32_T>(threadId % 8ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 8ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 810)) &&
                            (static_cast<int32_T>(i1 < 8)))) &&
      (static_cast<int32_T>(i < 8))) {
    varargout_3_Data[(i + (i1 << 3)) + (p << 6)] =
        output[(i1 + (i << 3)) + (p << 6)];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel5(
    const real32_T output[2304], real32_T varargout_4_Data[2304])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 8ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 8ULL;
  i1 = static_cast<int32_T>(threadId % 8ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 8ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 36)) &&
                            (static_cast<int32_T>(i1 < 8)))) &&
      (static_cast<int32_T>(i < 8))) {
    varargout_4_Data[(i + (i1 << 3)) + (p << 6)] =
        output[(i1 + (i << 3)) + (p << 6)];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel6(
    const real32_T output[207360], real32_T varargout_5_Data[207360])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 16ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 16ULL;
  i1 = static_cast<int32_T>(threadId % 16ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 16ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 810)) &&
                            (static_cast<int32_T>(i1 < 16)))) &&
      (static_cast<int32_T>(i < 16))) {
    varargout_5_Data[(i + (i1 << 4)) + (p << 8)] =
        output[(i1 + (i << 4)) + (p << 8)];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel7(
    const real32_T output[9216], real32_T varargout_6_Data[9216])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 16ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 16ULL;
  i1 = static_cast<int32_T>(threadId % 16ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 16ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 36)) &&
                            (static_cast<int32_T>(i1 < 16)))) &&
      (static_cast<int32_T>(i < 16))) {
    varargout_6_Data[(i + (i1 << 4)) + (p << 8)] =
        output[(i1 + (i << 4)) + (p << 8)];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel8(
    const real32_T output[829440], real32_T varargout_7_Data[829440])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 32ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 32ULL;
  i1 = static_cast<int32_T>(threadId % 32ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 32ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 810)) &&
                            (static_cast<int32_T>(i1 < 32)))) &&
      (static_cast<int32_T>(i < 32))) {
    varargout_7_Data[(i + (i1 << 5)) + (p << 10)] =
        output[(i1 + (i << 5)) + (p << 10)];
  }
}

static __global__ __launch_bounds__(512, 1) void dlnetwork_predict_kernel9(
    const real32_T output[36864], real32_T varargout_8_Data[36864])
{
  uint64_T threadId;
  int32_T i;
  int32_T i1;
  int32_T p;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId % 32ULL);
  threadId = (threadId - static_cast<uint64_T>(i)) / 32ULL;
  i1 = static_cast<int32_T>(threadId % 32ULL);
  threadId = (threadId - static_cast<uint64_T>(i1)) / 32ULL;
  p = static_cast<int32_T>(threadId);
  if ((static_cast<int32_T>((static_cast<int32_T>(p < 36)) &&
                            (static_cast<int32_T>(i1 < 32)))) &&
      (static_cast<int32_T>(i < 32))) {
    varargout_8_Data[(i + (i1 << 5)) + (p << 10)] =
        output[(i1 + (i << 5)) + (p << 10)];
  }
}

namespace coder {
void dlnetwork_predict(
    efficientDetD0_coco0_0 *obj, const k_dlarray *varargin_1,
    real32_T varargout_1_Data[12960], real32_T varargout_2_Data[576],
    real32_T varargout_3_Data[51840], real32_T varargout_4_Data[2304],
    real32_T varargout_5_Data[207360], real32_T varargout_6_Data[9216],
    real32_T varargout_7_Data[829440], real32_T varargout_8_Data[36864],
    real32_T varargout_9_Data[3317760], real32_T varargout_10_Data[147456])
{
  efficientDetD0_coco0_0 *gpu_obj;
  cell_wrap_6(*gpu_inputDataT)[1];
  k_dlarray *gpu_varargin_1;
  real32_T(*gpu_varargout_9_Data)[3317760];
  real32_T(*i_gpu_output)[3317760];
  real32_T(*g_gpu_output)[829440];
  real32_T(*gpu_varargout_7_Data)[829440];
  real32_T(*e_gpu_output)[207360];
  real32_T(*gpu_varargout_5_Data)[207360];
  real32_T(*gpu_varargout_10_Data)[147456];
  real32_T(*j_gpu_output)[147456];
  real32_T(*c_gpu_output)[51840];
  real32_T(*gpu_varargout_3_Data)[51840];
  real32_T(*gpu_varargout_8_Data)[36864];
  real32_T(*h_gpu_output)[36864];
  real32_T(*gpu_output)[12960];
  real32_T(*gpu_varargout_1_Data)[12960];
  real32_T(*f_gpu_output)[9216];
  real32_T(*gpu_varargout_6_Data)[9216];
  real32_T(*d_gpu_output)[2304];
  real32_T(*gpu_varargout_4_Data)[2304];
  real32_T(*b_gpu_output)[576];
  real32_T(*gpu_varargout_2_Data)[576];
  hipMalloc(&gpu_varargout_10_Data, 589824ULL);
  hipMalloc(&gpu_varargout_9_Data, 13271040ULL);
  hipMalloc(&gpu_varargout_8_Data, 147456ULL);
  hipMalloc(&gpu_varargout_7_Data, 3317760ULL);
  hipMalloc(&gpu_varargout_6_Data, 36864ULL);
  hipMalloc(&gpu_varargout_5_Data, 829440ULL);
  hipMalloc(&gpu_varargout_4_Data, 9216ULL);
  hipMalloc(&gpu_varargout_3_Data, 207360ULL);
  hipMalloc(&gpu_varargout_2_Data, 2304ULL);
  hipMalloc(&gpu_varargout_1_Data, 51840ULL);
  hipMalloc(&j_gpu_output, 589824ULL);
  hipMalloc(&i_gpu_output, 13271040ULL);
  hipMalloc(&h_gpu_output, 147456ULL);
  hipMalloc(&g_gpu_output, 3317760ULL);
  hipMalloc(&f_gpu_output, 36864ULL);
  hipMalloc(&e_gpu_output, 829440ULL);
  hipMalloc(&d_gpu_output, 9216ULL);
  hipMalloc(&c_gpu_output, 207360ULL);
  hipMalloc(&b_gpu_output, 2304ULL);
  hipMalloc(&gpu_output, 51840ULL);
  hipMalloc(&gpu_obj, 1ULL);
  hipMalloc(&gpu_inputDataT, 3145728ULL);
  hipMalloc(&gpu_varargin_1, 3145728ULL);
  hipMemcpy(gpu_varargin_1, varargin_1, 3145728ULL, hipMemcpyHostToDevice);
  dlnetwork_predict_kernel1<<<dim3(1536U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      gpu_varargin_1, *gpu_inputDataT);
  hipMemcpy(gpu_obj, obj, 1ULL, hipMemcpyHostToDevice);
  hipMemcpy(obj->getInputDataPointer(0), (*gpu_inputDataT)[0].f1,
             obj->getLayerOutputSize(0, 0), hipMemcpyDeviceToDevice);
  try {
    obj->activations(598);
  } catch (std::runtime_error const &err) {
    checkRunTimeError(err.what(), __FILE__, __LINE__);
  } catch (...) {
    checkRunTimeError("", __FILE__, __LINE__);
  }
  hipMemcpy(*gpu_output, obj->getLayerOutput(463, 0),
             obj->getLayerOutputSize(463, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*b_gpu_output, obj->getLayerOutput(478, 0),
             obj->getLayerOutputSize(478, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*c_gpu_output, obj->getLayerOutput(493, 0),
             obj->getLayerOutputSize(493, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*d_gpu_output, obj->getLayerOutput(508, 0),
             obj->getLayerOutputSize(508, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*e_gpu_output, obj->getLayerOutput(523, 0),
             obj->getLayerOutputSize(523, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*f_gpu_output, obj->getLayerOutput(538, 0),
             obj->getLayerOutputSize(538, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*g_gpu_output, obj->getLayerOutput(553, 0),
             obj->getLayerOutputSize(553, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*h_gpu_output, obj->getLayerOutput(568, 0),
             obj->getLayerOutputSize(568, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*i_gpu_output, obj->getLayerOutput(583, 0),
             obj->getLayerOutputSize(583, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(*j_gpu_output, obj->getLayerOutput(598, 0),
             obj->getLayerOutputSize(598, 0), hipMemcpyDeviceToDevice);
  hipMemcpy(obj, gpu_obj, 1ULL, hipMemcpyDeviceToHost);
  dlnetwork_predict_kernel2<<<dim3(26U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_output, *gpu_varargout_1_Data);
  dlnetwork_predict_kernel3<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *b_gpu_output, *gpu_varargout_2_Data);
  dlnetwork_predict_kernel4<<<dim3(102U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *c_gpu_output, *gpu_varargout_3_Data);
  dlnetwork_predict_kernel5<<<dim3(5U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *d_gpu_output, *gpu_varargout_4_Data);
  dlnetwork_predict_kernel6<<<dim3(405U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *e_gpu_output, *gpu_varargout_5_Data);
  dlnetwork_predict_kernel7<<<dim3(18U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *f_gpu_output, *gpu_varargout_6_Data);
  dlnetwork_predict_kernel8<<<dim3(1620U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *g_gpu_output, *gpu_varargout_7_Data);
  dlnetwork_predict_kernel9<<<dim3(72U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *h_gpu_output, *gpu_varargout_8_Data);
  dlnetwork_predict_kernel10<<<dim3(6480U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *i_gpu_output, *gpu_varargout_9_Data);
  dlnetwork_predict_kernel11<<<dim3(288U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *j_gpu_output, *gpu_varargout_10_Data);
  hipMemcpy(varargout_1_Data, *gpu_varargout_1_Data, 51840ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(varargout_2_Data, *gpu_varargout_2_Data, 2304ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(varargout_3_Data, *gpu_varargout_3_Data, 207360ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(varargout_4_Data, *gpu_varargout_4_Data, 9216ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(varargout_5_Data, *gpu_varargout_5_Data, 829440ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(varargout_6_Data, *gpu_varargout_6_Data, 36864ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(varargout_7_Data, *gpu_varargout_7_Data, 3317760ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(varargout_8_Data, *gpu_varargout_8_Data, 147456ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(varargout_9_Data, *gpu_varargout_9_Data, 13271040ULL,
             hipMemcpyDeviceToHost);
  hipMemcpy(varargout_10_Data, *gpu_varargout_10_Data, 589824ULL,
             hipMemcpyDeviceToHost);
  hipFree(gpu_varargin_1);
  hipFree(*gpu_inputDataT);
  hipFree(gpu_obj);
  hipFree(*gpu_output);
  hipFree(*b_gpu_output);
  hipFree(*c_gpu_output);
  hipFree(*d_gpu_output);
  hipFree(*e_gpu_output);
  hipFree(*f_gpu_output);
  hipFree(*g_gpu_output);
  hipFree(*h_gpu_output);
  hipFree(*i_gpu_output);
  hipFree(*j_gpu_output);
  hipFree(*gpu_varargout_1_Data);
  hipFree(*gpu_varargout_2_Data);
  hipFree(*gpu_varargout_3_Data);
  hipFree(*gpu_varargout_4_Data);
  hipFree(*gpu_varargout_5_Data);
  hipFree(*gpu_varargout_6_Data);
  hipFree(*gpu_varargout_7_Data);
  hipFree(*gpu_varargout_8_Data);
  hipFree(*gpu_varargout_9_Data);
  hipFree(*gpu_varargout_10_Data);
}

} // namespace coder

// End of code generation (predict.cu)
