//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// efficientDetD0_predict_terminate.cu
//
// Code generation for function 'efficientDetD0_predict_terminate'
//

// Include files
#include "efficientDetD0_predict_terminate.h"
#include "_coder_efficientDetD0_predict_mex.h"
#include "efficientDetD0_predict.h"
#include "efficientDetD0_predict_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void efficientDetD0_predict_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  efficientDetD0_predict_free();
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
  hipFree(*xq_gpu_clone);
  hipFree(*b_xq_gpu_clone);
  hipFree(*c_xq_gpu_clone);
  hipFree(*d_xq_gpu_clone);
}

void efficientDetD0_predict_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (efficientDetD0_predict_terminate.cu)
