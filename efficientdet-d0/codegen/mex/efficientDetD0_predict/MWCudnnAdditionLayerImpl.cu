#include "hip/hip_runtime.h"
#include "MWCudnnAdditionLayerImpl.hpp"
#include "MWCudnnCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include <cstdarg>
#include <cassert>
 namespace MWCudnnTarget { MWAdditionLayerImpl::MWAdditionLayerImpl(MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) { 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); } 
MWAdditionLayerImpl::~MWAdditionLayerImpl() { } void 
MWAdditionLayerImpl::propagateSize() { MWCNNLayer* AdditionLayer = getLayer(); 
MWTensorBase* opTensor = AdditionLayer->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); setDescriptor<float>(*desc, 
static_cast<MWTensor<float>*>(opTensor)); } void __global__ 
__launch_bounds__(1024) addImpl(float* in1, float* in2, float* out, size_t 
maxElems) { size_t i = blockDim.x * blockIdx.x + threadIdx.x; for (; i < 
maxElems; i += size_t(blockDim.x * gridDim.x)) { out[i] = in1[i] + in2[i]; } } 
void MWAdditionLayerImpl::predict() { MWCNNLayer* AdditionLayer = getLayer(); 
MWTensorBase* ipTensorBase = AdditionLayer->getInputTensor(0); MWTensorBase* 
opTensorBase = AdditionLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); int dJcdBfQQLhIAYHPxwQeg = 
ipTensor->getNumElements(); hipMemcpy(opTensor->getData(), 
ipTensor->getData(), dJcdBfQQLhIAYHPxwQeg * sizeof(float), 
hipMemcpyDeviceToDevice); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); for (int k = 1; k 
< AdditionLayer->getNumInputs(); k++) { hipdnnTensorDescriptor_t ipDesc = 
MWCNNLayerImpl::getCuDNNDescriptor(AdditionLayer->getInputTensor(k)); 
hipdnnAddTensor(*cQBKlCKXxecGPJrXBXdk->getCudnnHandle(), getOnePtr(), ipDesc, 
static_cast<MWTensor<float>*>(AdditionLayer->getInputTensor(k))->getData(), 
getOnePtr(), *desc, opTensor->getData()); } } void 
MWAdditionLayerImpl::cleanup() { } } 